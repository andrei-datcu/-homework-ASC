#include "hip/hip_runtime.h"
/* Nume: Datcu Andrei Daniel
   Grupa: 331CC
   Tema 4 ASC
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P);


__device__ inline bool in_bounds(int row, int col, int height, int width){
    return row >= 0 && row < height && col >=0 && col < width;
}

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{

    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (!in_bounds(row, col, N.height, N.width))
        return;//Daca nu fac parte din matrice nu am ce calcula

    for (int i = row - KERNEL_SIZE / 2, ci = 0; i <=  row + KERNEL_SIZE / 2; ++i, ++ci)
        for (int j = col - KERNEL_SIZE / 2, cj = 0; j <=  col + KERNEL_SIZE /2; ++j, ++cj)
            Cvalue += M[ci][cj] * (in_bounds(i, j, N.height, N.width) ? N[i][j] : 0);

    P[row][col] = Cvalue;
}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{

    unsigned int startOffsetY = blockIdx.y * BLOCK_SIZE;
    unsigned int startOffsetX = blockIdx.x * BLOCK_SIZE;

    if (startOffsetY >= N.height || startOffsetX  >= N.width)
        return; //Acest block nu face parte din imagine

    //Dimensiunile actuale ale blocului, la margini - cand nu mai e loc pentru
    //block full
    int actualBSX = min(N.width - startOffsetX, (unsigned int)BLOCK_SIZE);
    int actualBSY = min(N.height - startOffsetY, (unsigned int)BLOCK_SIZE);

    //liniile si coloanele relative la bloc
    int col = threadIdx.x, row = threadIdx.y;

    __shared__ float Ms[KERNEL_SIZE][KERNEL_SIZE];

    //Matricea M va fi copiata in memoria shared de KERNEL_SIZE * KERNEL_SIZE
    //threaduri incepand de la threadul KERNEL_SIZE/2, KERNEL_SIZE/2

    if (row >= KERNEL_SIZE / 2 && row < KERNEL_SIZE + KERNEL_SIZE / 2 &&
            col >= KERNEL_SIZE / 2 && col < KERNEL_SIZE + KERNEL_SIZE / 2)
        Ms[row - KERNEL_SIZE / 2][col - KERNEL_SIZE / 2] =
            M[row - KERNEL_SIZE / 2][col - KERNEL_SIZE / 2];

    if (col >= actualBSX || row >= actualBSY)
        return;// threadul din acest bloc nu corespunde unui pixel din imagine

    __shared__ float Ns[BLOCK_SIZE+ 2 * (KERNEL_SIZE / 2)]
                        [BLOCK_SIZE + 2 * (KERNEL_SIZE / 2)];

    //Calculam indicii relativ la coltul dreapta sus al imaginii
    int fullCol = startOffsetX + col, fullRow = startOffsetY + row;

    // Aduc datele in memoria shared

    // Fiecare thread isi aduce pixelul corespunzator in memoria shared
    Ns[row + KERNEL_SIZE / 2][col + KERNEL_SIZE / 2] = N[fullRow][fullCol];

    // Threadurile de pe randurile din margine aduc
    //fiecare cate unul in plus pentru bordare sus
    if (actualBSY >= KERNEL_SIZE / 2){
        if (row < KERNEL_SIZE / 2)
            Ns[row][col + KERNEL_SIZE / 2] =
                (fullRow - KERNEL_SIZE / 2 >= 0) ?
                    N[fullRow - KERNEL_SIZE / 2][fullCol] : 0;
    }
    else //Daca nu sunt suficiente pe margine, unul bordeaza tot
        if (row == 0)
            for (int i = 0; i < KERNEL_SIZE / 2; ++i)
                Ns[i][col + KERNEL_SIZE / 2] =
                    (fullRow + i - KERNEL_SIZE / 2 >= 0) ?
                        N[fullRow + i - KERNEL_SIZE / 2][fullCol] : 0;

    // Threadurile de pe coloanele din margine aduc fiecare cate un pixel
    //plus pentru bordare la stanga
    if (actualBSX >= KERNEL_SIZE / 2){
        if (col < KERNEL_SIZE / 2)
            Ns[row + KERNEL_SIZE / 2][col] =
                (fullCol - KERNEL_SIZE / 2 >= 0) ?
                    N[fullRow][fullCol - KERNEL_SIZE / 2] : 0;
    }
    else if (col == 0) // Daca nu sunt suficiente pe margine unu bordeaza tot
        for (int j = 0; j < KERNEL_SIZE / 2; ++j)
            Ns[row + KERNEL_SIZE / 2][j] =
                (fullCol +j - KERNEL_SIZE / 2 >= 0) ?
                    N[fullRow][fullCol +j - KERNEL_SIZE / 2] : 0;


    // Threadurile din coltul stanga sus bordeaza stanga sus
    if (col == 0 && row == 0)
        for (int dy = - KERNEL_SIZE / 2, sy = 0; dy < 0; ++dy, ++sy)
            for (int dx = - KERNEL_SIZE /2, sx = 0; dx < 0; ++dx, ++sx)
                if (fullRow + dy >= 0 && fullCol + dx >= 0)
                    Ns[sy][sx] = N[fullRow + dy][fullCol + dx];
                else
                    Ns[sy][sx] = 0;

    // Threadurile din coltul stanga jos bordeaza stanga jos
    if (col == 0 && row == actualBSY - 1)
        for (int dy = 1, sy = actualBSY + KERNEL_SIZE / 2;
             dy <= KERNEL_SIZE / 2; ++dy, ++sy)
            for (int dx = - KERNEL_SIZE /2, sx = 0; dx < 0; ++dx, ++sx)
                if (fullRow + dy < N.height && fullCol + dx >= 0)
                    Ns[sy][sx] = N[fullRow + dy][fullCol + dx];
                else
                    Ns[sy][sx] = 0;

    //Bordare jos la fel ca mai sus
    if (actualBSY >= KERNEL_SIZE / 2){
        if (row >= actualBSY - KERNEL_SIZE / 2)
            Ns[row + 2 * (KERNEL_SIZE/2)][col + KERNEL_SIZE / 2] =
                (fullRow + KERNEL_SIZE / 2 < N.height) ?
                    N[fullRow + KERNEL_SIZE / 2][fullCol] : 0;
    }
    else if (row == actualBSY - 1)
        for (int i = 0; i < KERNEL_SIZE / 2; ++i)
            Ns[i + actualBSY + KERNEL_SIZE / 2][col + KERNEL_SIZE / 2] =
                (fullRow + i + 1 < N.height) ? N[fullRow + i + 1][fullCol] : 0;


    //Bordare dreapta la fel ca mai sus
    if (actualBSX >= KERNEL_SIZE / 2){
        if (col >= actualBSX - KERNEL_SIZE / 2)
            Ns[row + KERNEL_SIZE / 2][col + 2 * (KERNEL_SIZE / 2)] =
                (fullCol + KERNEL_SIZE / 2 < N.width) ?
                    N[fullRow][fullCol + KERNEL_SIZE / 2] : 0;
    }
    else if (col == actualBSX - 1)
        for (int j = 0; j < KERNEL_SIZE / 2; ++j)
            Ns[row + KERNEL_SIZE / 2][actualBSX + j + KERNEL_SIZE / 2] =
                (fullCol + j + 1 < N.width) ? N[fullRow][fullCol + j + 1] : 0;

    //Bordare dreapta sus
    if (col == actualBSX - 1 && row == 0)
        for (int dy = - KERNEL_SIZE / 2, sy = 0; dy < 0; ++dy, ++sy)
            for (int dx = 1, sx = actualBSX + KERNEL_SIZE / 2;
                    dx <= KERNEL_SIZE / 2; ++dx, ++sx)
                if (fullRow + dy >= 0 && fullCol + dx < N.width)
                    Ns[sy][sx] = N[fullRow + dy][fullCol + dx];
                else
                    Ns[sy][sx] = 0;

    //Bordare stanga sus
    if (col == actualBSX - 1 && row == actualBSY - 1)
        for (int dy = 1, sy = actualBSY + KERNEL_SIZE / 2;
                dy <= KERNEL_SIZE / 2; ++dy, ++sy)
            for (int dx = 1, sx = actualBSX + KERNEL_SIZE / 2;
                    dx <= KERNEL_SIZE / 2; ++dx, ++sx)
                if (fullRow + dy < N.height && fullCol + dx < N.width)
                    Ns[sy][sx] = N[fullRow + dy][fullCol + dx];
                else
                    Ns[sy][sx] = 0;

    __syncthreads();

    float C = 0;

    for (int dy = row, cy = 0; dy < row + KERNEL_SIZE; ++dy, ++cy)
        for (int dx = col, cx = 0; dx < col + KERNEL_SIZE; ++dx, ++cx)
            C += Ms[cy][cx] * Ns[dy][dx];

    //Punem valorea finala a pixelului in imagine
    P[fullRow][fullCol] = C;
}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++)
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    int width = 0, height = 0;
    FILE *f, *out;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);
    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    fscanf(f, "%d%d", &width, &height);
    Matrix M;//kernel de pe host
    Matrix N;//matrice inițială de pe host
    Matrix P;//rezultat fără memorie partajată calculat pe GPU
    Matrix PS;//rezultatul cu memorie partajată calculat pe GPU

    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);

    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);


    // M * N pe device
    ConvolutionOnDevice(M, N, P);

    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS);

    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.width, P.height);
    struct timeval t1, t2;

    gettimeofday(&t1, NULL);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    gettimeofday(&t2, NULL);

    fprintf (stderr, "Timp execuție cpu: %lf ms\n",(t2.tv_sec - (t1).tv_sec) *
             1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0);

    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(out, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(out, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");

    // Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //: alocați matricele de pe device

    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    //: copiați datele de pe host (M, N) pe device (MD, Nd)
    hipMemcpy(Md.elements, M.elements, M.sizeInBytes(), hipMemcpyHostToDevice);
    hipMemcpy(Nd.elements, N.elements, N.sizeInBytes(), hipMemcpyHostToDevice);

    //: setați configurația de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N.width / dimBlock.x + 1, N.height / dimBlock.y + 1);

    sdkStartTimer(&kernelTime);
    //: lansați în execuție kernelul
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    fprintf (stderr, "Timp execuție kernel: %f ms\n", sdkGetTimerValue(&kernelTime));
    //: copiaţi rezultatul pe host
    hipMemcpy(P.elements, Pd.elements, P.sizeInBytes(), hipMemcpyDeviceToHost);
    //: eliberați memoria matricelor de pe device
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    //: alocați matricele de pe device
    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    //: copiați datele de pe host (M, N) pe device (MD, Nd)
    hipMemcpy(Md.elements, M.elements, M.sizeInBytes(), hipMemcpyHostToDevice);
    hipMemcpy(Nd.elements, N.elements, N.sizeInBytes(), hipMemcpyHostToDevice);

    //: setați configurația de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N.width / dimBlock.x + 1, N.height / dimBlock.y + 1);

    sdkStartTimer(&kernelTime);
    //: lansați în execuție kernelul
    ConvolutionKernelShared<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    fprintf (stderr, "Timp execuție kernel cu memorie partajată: %f ms\n",
             sdkGetTimerValue(&kernelTime));
    //: copiaţi rezultatul pe host
    hipMemcpy(P.elements, Pd.elements, P.sizeInBytes(), hipMemcpyDeviceToHost);
    //: eliberați memoria matricelor de pe device

    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


// Alocă o matrice de dimensiune height*width pe device
Matrix AllocateDeviceMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    hipMalloc((void**)&M.elements, size * sizeof(float));
    return M;
}

// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = (float*) malloc(size*sizeof(float));
    return M;
}

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}
